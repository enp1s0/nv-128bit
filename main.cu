
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <memory>

constexpr std::size_t N = 1 << 28;
constexpr std::size_t mem_N = N * 4;
constexpr std::size_t num_threads = 1 << 8;
constexpr std::size_t test_count = 1 << 10;

__global__ void read_global(float* const dst, const float* const src){
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

	const auto tmp0 = src[tid * 4 + 0];
	const auto tmp1 = src[tid * 4 + 1];
	const auto tmp2 = src[tid * 4 + 2];
	const auto tmp3 = src[tid * 4 + 3];

	dst[tid] = tmp0 * tmp1 * tmp2* tmp3;
}

__global__ void read_global_128(float* const dst, const float* const src){
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;

	const auto tmp = reinterpret_cast<const float4*>(src);

	dst[tid] = tmp->x * tmp->y * tmp->z * tmp->z;
}

template <class Func>
double get_elapsed_time(Func func){
	const auto start = std::chrono::system_clock::now();
	func();
	hipDeviceSynchronize();
	const auto end = std::chrono::system_clock::now();
	return std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count() / 1000.0;
}

template <class T>
auto get_device_uptr(const std::size_t N){
	struct deleter{
		void operator()(T* const ptr){hipFree(ptr);};
	};
	T* ptr;
	hipMalloc((void**)&ptr, sizeof(T) * N);
	return std::unique_ptr<T, deleter>{ptr};
}

int main(){
	{
		auto srt_uptr = get_device_uptr<float>(mem_N);
		auto dst_uptr = get_device_uptr<float>(N);
		const auto elapsed_time = get_elapsed_time(
					[&srt_uptr, &dst_uptr](){
						for(std::size_t c = 0; c < test_count; c++) read_global_128<<<(N / num_threads), num_threads>>>(srt_uptr.get(), dst_uptr.get());
					});
		std::cout<<"    128bit read : "<<elapsed_time<<" [s]"<<std::endl;
	}
	{
		auto srt_uptr = get_device_uptr<float>(mem_N);
		auto dst_uptr = get_device_uptr<float>(N);
		const auto elapsed_time = get_elapsed_time(
					[&srt_uptr, &dst_uptr](){
						for(std::size_t c = 0; c < test_count; c++) read_global<<<(N / num_threads), num_threads>>>(srt_uptr.get(), dst_uptr.get());
					});
		std::cout<<" 32bit x 4 read : "<<elapsed_time<<" [s]"<<std::endl;
	}
}
